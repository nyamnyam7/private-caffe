#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"


namespace caffe {

template <typename Dtype>
__global__ void NMSForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height, const int width, 
    const int kernel_h, const int kernel_w,
    Dtype* top_data,
    Dtype* mask) {
  // Iterate over top_data
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % width;
    int h = (index / width) % height;
    int n = index / width / height / channels;
    int c = (index / width / height) % channels;

    int jstart = w - kernel_w;
    int jend = w + kernel_w+1;
    int istart = h - kernel_h;
    int iend = h + kernel_h+1;
    
    Dtype curval = bottom_data[index];
    bool is_maximum = true;
    // extremely inefficient implementation of non-maximum suprression
    const Dtype* rel = bottom_data + (n * channels + c) * height * width;
    if (jstart > 0 && jend <= width && istart > 0 && iend <= height)
    {
        for (int i=istart; i<iend; i++){
            for (int j=jstart; j<jend; j++){
                if ( rel[i * width + j] > curval ) is_maximum = false;
            }
        }
    }
    else is_maximum = false;

    if (is_maximum) {
        top_data[index] = bottom_data[index];
        mask[index] = 1.0;
    }
    else
    {
        top_data[index] = 0.0;
        mask[index] = 0.0;
    }
  }
}


template <typename Dtype>
void NMSLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  Dtype* mask = NULL;

  if (use_top_mask) {
    mask = top[1]->mutable_gpu_data();
  } else {
    mask = mask_.mutable_gpu_data();
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  NMSForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->num(), channels_,
      height_, width_, kernel_h_, kernel_w_,
      top_data, mask);
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void NMSBackward(const int nthreads,
    const Dtype* top_diff, const Dtype* mask, Dtype* bottom_diff) {
  // Iterate over bottom_diff
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    if (mask[index] > 0) bottom_diff[index] = top_diff[index];
    else bottom_diff[index] = 0.0;
  }
}


template <typename Dtype>
void NMSLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();

  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  const Dtype* mask = NULL;
  if (use_top_mask) {
    mask = top[1]->gpu_data();
  } else {
    mask = mask_.gpu_data();
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  NMSBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, mask, bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(NMSLayer);


}  // namespace caffe
