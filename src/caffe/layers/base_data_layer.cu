#include <vector>

#include "caffe/data_layers.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

template <typename Dtype>
void BaseExtendedPrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  ExtendedBatch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  for (int i=0; i<top.size(); i++) {
    top[i]->ReshapeLike(*batch->data_[i]);
    // Copy the data
    caffe_copy(batch->data_[i]->count(), batch->data_[i]->gpu_data(),
        top[i]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}


INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(BaseExtendedPrefetchingDataLayer);

}  // namespace caffe
